#include "hip/hip_runtime.h"
#include <stdio.h> // for CUDA's printf

// #include "lightweight_test_label.hpp"
#include "test_utils.hpp"
#include <strf.hpp>
#include <sstream>

// Note: There are adaptations of, say, std::span for use with CUDA (= I adapted it...).
// But we want to avoid dependency clutter here, so let's just stick to the basics.


__global__ void kernel_using_cstr_writer(strf::cstr_writer::result* write_result, char* buffer, std::size_t buffer_size)
{
//  int global_thread_id = threadIdx.x + blockIdx.x * blockDim.x;
//  strf::snprintf(buf, "Thread %d says: Hello %s\n", global_thread_id, "world.");
//  printf("Thread %3d says: Hello %s\n", global_thread_id, "world.");
  strf::basic_cstr_writer<char> sw(buffer, buffer_size);
  write(sw, "Hello");
  write(sw, " world");
  *write_result = sw.finish();

//  if (not write_result->truncated) {
//	  printf("[%s kernel, thread %03d] Finalized string is: \"%s\"\n", __FUNCTION__, global_thread_id, buffer);
//  }
//  else {
//	  printf("[%s kernel, thread %03d] Finalized string is: \"%11s\"\n", __FUNCTION__, global_thread_id, buffer);
//  }
}

//__global__ void kernel_using_cstr_to(char* buffer, std::size_t buffer_size)
//{
//  int global_thread_id = threadIdx.x + blockIdx.x * blockDim.x;
//  auto printer = strf::to(buffer, buffer_size);
//  printer ( "Hello", ' ', "world, from thread ", global_thread_id );
//}



// Ugly, no-good error-checking.
#define ensure_cuda_success(ans) { ensure_cuda_success_((ans), __FILE__, __LINE__); }

inline void ensure_cuda_success_(hipError_t status, const char *file, int line, bool abort=true)
{
  BOOST_TEST_EQ(status, hipSuccess);
  if (abort and (status != hipSuccess)) {
    BOOST_ERROR(hipGetErrorString(status));
    exit(boost::report_errors());
  }
}


void test_cstr_writer()
{
	struct args {
		strf::cstr_writer::result write_result;
		char buffer[50];
	};
	const std::size_t buffer_size { std::strlen("Hello world") + 1 }; // Enough for "Hello world" with the trailing '\0'.
	struct args* device_side_args;
	ensure_cuda_success(hipMalloc(&device_side_args, sizeof(struct args)));
	ensure_cuda_success(hipMemset(device_side_args, 0, sizeof(struct args)));

	int threads_per_block { 1 };
	int blocks_in_grid { 1 };
		// We could theoretically have multiple threads in multiple blocks run this, but
		// it shouldn't really matter.
	kernel_using_cstr_writer<<<threads_per_block, blocks_in_grid>>>(
		&(device_side_args->write_result),
		&(device_side_args->buffer[0]),
		buffer_size);
	ensure_cuda_success(hipGetLastError());
	ensure_cuda_success(hipDeviceSynchronize());
	args host_side_args;
	ensure_cuda_success(hipMemcpy(&host_side_args, device_side_args, sizeof(struct args), hipMemcpyDeviceToHost));
    BOOST_TEST_EQ(host_side_args.write_result.truncated, false);
	BOOST_TEST_EQ(host_side_args.write_result.ptr, &(device_side_args->buffer[0]) + std::strlen("Hello world"));
	if (host_side_args.write_result.ptr == &(device_side_args->buffer[0])) {
		BOOST_TEST_EQ(strncmp(host_side_args.write_result.ptr, host_side_args.buffer, buffer_size), 0);
	}
}

/*
void test_cstr_to()
{
	char* device_side_buffer;
	const std::size_t buffer_size { 100 }; // More than enough for "Hello world from thread XYZ"
	ensure_cuda_success( hipMalloc(&device_side_buffer, buffer_size) );
	ensure_cuda_success( hipMemset(device_side_buffer, 0, buffer_size) );

	int threads_per_block { 1 };
	int blocks_in_grid { 1 };
		// We could theoretically have multiple threads in multiple blocks run this, but
		// it shouldn't really matter.
	kernel_using_cstr_to<<<threads_per_block, blocks_in_grid>>>(device_side_buffer, buffer_size);
	ensure_cuda_success(hipGetLastError());
	ensure_cuda_success(hipDeviceSynchronize());
	char host_side_buffer[buffer_size];
	ensure_cuda_success(hipMemcpy(&host_side_buffer, device_side_buffer, buffer_size , hipMemcpyDeviceToHost));
	std::stringstream expected;
	expected << "Hello" << ' ' << "world, from thread " << 1;
	BOOST_TEST_EQ(strncmp(host_side_buffer, expected.str().c_str(), buffer_size), 0);
}
*/

void cstr_to_sanity_check()
{
	const std::size_t buffer_size { 100 }; // More than enough for "Hello world from thread XYZ"
	char buffer[buffer_size];
	std::fill_n(buffer, sizeof(buffer), 0);
    auto print_functor = strf::to(buffer, buffer_size);
    print_functor ( "Hello", ' ', "world, from thread ", 1 );
	std::stringstream expected;
	expected << "Hello" << ' ' << "world, from thread " << 1;
	BOOST_TEST_EQ(strncmp(buffer, expected.str().c_str(), buffer_size), 0);
}


int main(void)
{
    auto num_devices { 0 };
    auto status = hipGetDeviceCount(&num_devices);

    BOOST_TEST_EQ(status, hipSuccess);
    if (status != hipSuccess)
    {
    	std::stringstream ss;
    	ss << "hipGetDeviceCount failed: " << hipGetErrorString(status) <<  '\n';
        BOOST_ERROR(ss.str().c_str());
    }
    if (num_devices == 0) {
        std::cerr << "No devices - can't run this test\n";
        return boost::report_errors();
    }
	// TODO: Test basic_cstr_writer's with different character types
	test_cstr_writer();
	cstr_to_sanity_check();
//	test_cstr_to();

    hipDeviceReset();
    return boost::report_errors();
}
